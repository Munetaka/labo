/*
- ref
http://www.gdep.jp/page/view/251
- compile
nvcc -o 99bottles_cpu.exe 99bottles_cpu.cu
- run
./99botlles_cpu.exe
*/



#include <hip/hip_runtime.h>
#include <stdio.h>


int main(void)
{
    int b;

    for (b = 99; b >= 0; b--) {
        switch (b) {
        case 0:
            printf("no more bottles of beer on the wall, no more bottles of beer.\n");
            printf("go to the store and buy some more, 99 bottles of beer on the wall.\n");
            break;
        case 1:
            printf("1 bottle of beer on the wall, 1 bottle of beer.\n");
            printf("take one down and pass it around, no more bottles of beer on the wall\n");
            break;
        default:
            printf("%d bottles of beer on the wall, %d bottles of beer.\n", b, b);
            printf("take one down and pass it around, %d %s of beer on the wall.\n", b - 1, ((b - 1) > 1) ? "bottles" : "bottle");
            break;
        }
    }
    return 0;
}
