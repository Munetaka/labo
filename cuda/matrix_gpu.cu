#include "hip/hip_runtime.h"
/*
- ref
http://www.gdep.jp/page/view/251
http://hidemon-memo.blogspot.jp/2014/10/cuda.html
- compile
export PATH=/usr/local/cuda-8.0/bin:$PATH
nvcc -I /usr/local/cuda-8.0/samples/common/inc -o matrix_gpu.exe matrix_gpu.cu
- run
./matrix_gpu.exe
*/


#include <stdio.h>
// #include <malloc.h>
#include <stdlib.h>
// #include <time.h>
// #include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>


#define MATRIX_SIZE 1024
#define BLOCK_SIZE 16


__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);


int main(int argc, char** argv)
{
    unsigned int matrixSize = sizeof(unsigned int) * MATRIX_SIZE * MATRIX_SIZE;

    int* hMatrixA;
    int* hMatrixB;
    int* hMatrixC;

    hMatrixA = (int*)malloc(matrixSize);
    hMatrixB = (int*)malloc(matrixSize);

    /* init */
    unsigned int col_idx, row_idx;
    for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++) {
        for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++) {
            hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % (MATRIX_SIZE * MATRIX_SIZE);
            hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % (MATRIX_SIZE * MATRIX_SIZE);
        }
    }

    /* device variable */
    int* dMatrixA;
    int* dMatrixB;
    int* dMatrixC;

    /* device memory */
    // cutilSafeCall(hipMalloc((void**)&dMatrixA, matrixSize));
    // cutilSafeCall(hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice));
    // cutilSafeCall(hipMalloc((void**)&dMatrixB, matrixSize));
    // cutilSafeCall(hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice));
    // cutilSafeCall(hipMalloc((void**)&dMatrixC, matrixSize));
    checkCudaErrors(hipMalloc((void**)&dMatrixA, matrixSize));
    checkCudaErrors(hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&dMatrixB, matrixSize));
    checkCudaErrors(hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&dMatrixC, matrixSize));

    /* block size & grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(MATRIX_SIZE / BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);

    /* start timer */
    // unsigned int timer = 0;
    // CUT_SAFE_CALL(cutCreateTimer(&timer));
    // CUT_SAFE_CALL(cutStartTimer(timer));
    hipEvent_t start;
    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, NULL)); // start

    /* start kernel */
    matrixMul<<<grid, block>>>(dMatrixA, dMatrixB, dMatrixC);
    hipDeviceSynchronize();

    /* secure mem & trancerate memory from device */
    hMatrixC = (int*)malloc(matrixSize);
    // cutilSafeCall(hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost));


    /* stop timer */
    // CUT_SAFE_CALL(cutStoptimer(timer));
    // printf("Processing time: %f (msec)\n", cutGetTimerValue(timer));
    // CUT_SAFE_CALL(cutDeleteTimer(timer));
    checkCudaErrors(hipEventRecord(stop, NULL));
    checkCudaErrors(hipEventSynchronize(stop));
    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    printf("Processing time: %f (msec)\n", msecTotal);

    /* release host, device memory */
    free(hMatrixA);
    free(hMatrixB);
    free(hMatrixC);
    // cutilSafeCall(hipFree(dMatrixA));
    // cutilSafeCall(hipFree(dMatrixB));
    // cutilSafeCall(hipFree(dMatrixC));
    checkCudaErrors(hipFree(dMatrixA));
    checkCudaErrors(hipFree(dMatrixB));
    checkCudaErrors(hipFree(dMatrixC));

    /* endroll */
    hipDeviceReset();
    exit(1);
}


__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC) {
    unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int scan_idx;
    unsigned int target = 0;

    for (scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
        target += inMatrixA[col_idx * MATRIX_SIZE + scan_idx] * inMatrixB[scan_idx * MATRIX_SIZE + row_idx];
        __syncthreads();
    }
    inMatrixC[col_idx * MATRIX_SIZE + row_idx] = target;
}
