/*
- ref
http://www.gdep.jp/page/view/251
- compile
nvcc -o matrix_cpu.exe matrix_cpu.cu
- run
./matrix_cpu.exe
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <time.h>


/* define n square matrix size */
#define MATRIX_SIZE 1024


int main(int argc, char** argv)
{
    unsigned int col_idx, row_idx, scan_idx;
    int* matA;
    int* matB;
    int* matC;

    time_t Start, Stop;

    matA = (int*)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    matB = (int*)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    matC = (int*)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);

    for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++) {
        for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++) {
            matA[col_idx * MATRIX_SIZE + row_idx] = rand() % (MATRIX_SIZE * MATRIX_SIZE);
            matB[col_idx * MATRIX_SIZE + row_idx] = rand() % (MATRIX_SIZE * MATRIX_SIZE);
            matC[col_idx * MATRIX_SIZE + row_idx] = 0;
        }
    }

    time(&Start);

    for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++) {
        for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++) {
            for (scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
                matC[col_idx + MATRIX_SIZE + row_idx] += matA[col_idx * MATRIX_SIZE + scan_idx] * matB[scan_idx * MATRIX_SIZE + row_idx];
            }
        }
    }

    time(&Stop);

    printf("Processing time: %f (sec)\n", (double)(Stop - Start));

    free(matA);
    free(matB);
    free(matC);

    return 0;
}
